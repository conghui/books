#include <stdio.h>
#include "info.h"

void getGeneralInfo(const hipDeviceProp_t *prop, int i) {
    printf("\n   --- General information for device %d ---\n", i);
    printf("Name:               %s\n", prop->name);
    printf("Compute capalibity: %d.%d\n", prop->major, prop->minor);
    printf("Clock rate:         %d\n", prop->clockRate);
    printf("Device copy overlap: ");

    if (prop->deviceOverlap) {
        printf("Enabled\n");
    }
    else {
        printf("Disabled\n");
    }

    printf("Kernel excition timeout: ");

    if (prop->kernelExecTimeoutEnabled) {
        printf("Enabled\n");
    }
    else {
        printf("Disabled\n");
    }
}

void getMemInfo(const hipDeviceProp_t *prop, int i) {
    printf("\n   --- Memory information for Device %d ---\n", i);
    printf("Total global mem:   %ld\n", prop->totalGlobalMem);
    printf("Total contant mem:  %ld\n", prop->totalConstMem);
    printf("Max mem pitch:      %ld\n", prop->memPitch);
    printf("Texture Alignment:  %ld\n", prop->textureAlignment);
}

void getMPInfo(const hipDeviceProp_t *prop, int i) {
    printf("\n   --- MP information for Device %d --\n", i);
    printf("Multiprocessor count:   %d\n", prop->multiProcessorCount);
    printf("Shared mem per mp:      %ld\n", prop->sharedMemPerBlock);
    printf("Register per mp:        %d\n", prop->regsPerBlock);
    printf("Threads in warp:        %d\n", prop->warpSize);
    printf("Max threads per block:  %d\n", prop->maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n",
            prop->maxThreadsDim[0],
            prop->maxThreadsDim[1],
            prop->maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n",
            prop->maxGridSize[0],
            prop->maxGridSize[1],
            prop->maxGridSize[2]);
}
