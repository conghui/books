#include <stdio.h>
#include "info.h"

int main(void) {
    int count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        getGeneralInfo(&prop, i);
        getMemInfo(&prop, i);
        getMPInfo(&prop, i);

    }
    return 0;
}
